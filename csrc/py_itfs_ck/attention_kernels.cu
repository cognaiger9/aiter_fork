// SPDX-License-Identifier: MIT
// Copyright (c) 2024, Advanced Micro Devices, Inc. All rights reserved.

#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include "py_itfs_common.h"
#include "ck_tile/ref/naive_attention.hpp"

torch::Tensor pa_fwd_naive(torch::Tensor &Q,            //   [num_seqs, num_heads, head_size]
                           torch::Tensor &K,            //   [num_blocks, num_kv_heads, head_size/x, block_size, x]
                                                        // or[num_batch, seqlen, num_kv_heads, head_size]
                           torch::Tensor &V,            //   [num_blocks, num_kv_heads, head_size, block_size]
                                                        // or[num_batch*seqlen, num_kv_heads, head_size]
                           torch::Tensor &block_tables, // [num_seqs, max_num_blocks_per_seq]
                           torch::Tensor &context_lens,
                           torch::Tensor &k_dequant_scales, // [num_heads, max_kv_tokens]
                           torch::Tensor &v_dequant_scales, // [num_heads, max_kv_tokens]
                           const int max_seq_len,
                           const int num_kv_heads,
                           const float scale_s,
                           const float scale_k,
                           const float scale_v,
                           const int block_size,
                           const int quant_algo, // 0: no quant, 1: per-token FP8 quant
                           std::optional<torch::Tensor> &out_)
{
    const at::cuda::OptionalCUDAGuard device_guard(device_of(Q));
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    TORCH_CHECK(block_tables.dtype() == torch::kInt32, "block_tables must be int32");
    TORCH_CHECK(context_lens.dtype() == torch::kInt32, "context_lens must be int32");
    torch::Tensor out = out_.value_or(torch::empty_like(Q));
    int batch = Q.size(0);
    int nhead = Q.size(1);
    int nhead_k = V.size(1);
    int hdim_q = Q.size(2);
    int hdim_v = V.size(2);
    int max_num_blocks_per_seq = block_tables.size(1);
    int max_kv_tokens = k_dequant_scales.numel() == 0 ? 0 : k_dequant_scales.size(1);

    ck_tile::naive_attention_fwd_traits naive_t;
    naive_t.q_type = torchDTypeToStr(Q.dtype());
    naive_t.k_type = torchDTypeToStr(K.dtype());
    naive_t.v_type = torchDTypeToStr(V.dtype());
    naive_t.o_type = torchDTypeToStr(out.dtype());
    naive_t.q_layout = "bhsd";
    naive_t.k_layout = "phdsx"; // TODO
    naive_t.v_layout = "phds";  // TODO
    naive_t.o_layout = "bhsd";
    naive_t.variation = 2; // decode variation
    naive_t.quant_algo = quant_algo;

    ck_tile::naive_attention_fwd_args naive_a;
    naive_a.q_ptr = Q.data_ptr();
    naive_a.k_ptr = K.data_ptr();
    naive_a.v_ptr = V.data_ptr();
    naive_a.o_ptr = out.data_ptr();
    naive_a.scale_s = scale_s;
    naive_a.context_len_ptr = context_lens.data_ptr(); // used when seqlen kv come from a pointer
    naive_a.page_table_ptr = block_tables.data_ptr();  // [batch, num_blocks] seqlen_kv is in different block(paged attn)
    naive_a.hdim = hdim_q;
    naive_a.hdim_v = hdim_v; // could be cross-attn, where V and Q/K hdim are different
    naive_a.batch_q = batch;
    naive_a.batch_kv = 1;           // decode case batch-kv always 1
    naive_a.batch_ratio_kv = batch; // batch_q / batch_kv
    naive_a.seqlen_q = 1;           // in decode case, this should be 1
    naive_a.seqlen_kv = 0;          // if context_len_ptr is not nullptr, ignore this field
    naive_a.nhead_q = nhead;
    naive_a.nhead_kv = nhead_k;
    naive_a.nhead_ratio_kv = naive_a.nhead_q / naive_a.nhead_kv; // nhead_q / nhead_kv
    naive_a.page_size = block_size;                              // if paged, the seqlen-kv for each block

    naive_a.kscale_ptr = k_dequant_scales.data_ptr();
    naive_a.vscale_ptr = v_dequant_scales.data_ptr();
    naive_a.max_pages_per_seq = max_num_blocks_per_seq;
    naive_a.max_kv_tokens = max_kv_tokens;

    ck_tile::stream_config naive_s{stream};

    naive_attention_fwd(naive_t, naive_a, naive_s);
    return out;
}