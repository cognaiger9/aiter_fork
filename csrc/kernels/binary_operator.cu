#include "hip/hip_runtime.h"
/*
 * Copyright © Advanced Micro Devices, Inc. All rights reserved.
 * Copyright (c) 2024, The vLLM team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include "hip_compat.h"
#include "dispatch_utils.h"
#include <torch/torch.h>

#ifdef USE_ROCM
#include <hip/hip_bf16.h>
typedef __hip_bfloat16 hip_bfloat16;
#else
#include <hip/hip_bf16.h>
#endif
#include <hip/hip_fp16.h>

namespace aiter
{
  template <typename T, typename Operation>
  inline __device__ T performOperation(T a, T b);

  template <typename Operation>
  torch::Tensor aten_compute(torch::Tensor &input, torch::Tensor &other);

  struct AddOp
  {
    template <typename T>
    inline __device__ static T apply(T a, T b) { return a + b; }

    static torch::Tensor compute(torch::Tensor &input, torch::Tensor &other)
    {
      return torch::add(input, other);
    }
  };

  struct SubOp
  {
    template <typename T>
    inline __device__ static T apply(T a, T b)
    {
      return a - b;
    }

    static torch::Tensor compute(torch::Tensor &input, torch::Tensor &other)
    {
      return torch::sub(input, other);
    }
  };

  struct MulOp
  {
    template <typename T>
    inline __device__ static T apply(T a, T b) { return a * b; }

    static torch::Tensor compute(torch::Tensor &input, torch::Tensor &other)
    {
      return torch::mul(input, other);
    }
  };

  struct DivOp
  {
    template <typename T>
    inline __device__ static T apply(T a, T b)
    {
      // assert(b == static_cast<T>(0));
      return a / b;
    }

    static torch::Tensor compute(torch::Tensor &input, torch::Tensor &other)
    {
      return torch::div(input, other);
    }
  };

  template <typename T, typename Operation, bool order_flag>
  inline __device__ T performOperation(T a, T b)
  {
    if constexpr (std::is_same_v<Operation, AddOp>)
    {
      return Operation::apply(a, b);
    }
    else if constexpr (std::is_same_v<Operation, SubOp>)
    {
      if constexpr (!order_flag)
      {
        return Operation::apply(b, a);
      }
      else
      {
        return Operation::apply(a, b);
      }
    }
    else if constexpr (std::is_same_v<Operation, MulOp>)
    {
      return Operation::apply(a, b);
    }
    else if constexpr (std::is_same_v<Operation, DivOp>)
    {
      if constexpr (!order_flag)
      {
        return Operation::apply(b, a);
      }
      else
      {
        return Operation::apply(a, b);
      }
    }
    else
    {
      static_assert(false, "Unsupported operation");
    }
  }
  template <typename Operation>
  torch::Tensor aten_compute(torch::Tensor &input, torch::Tensor &other)
  {
    if constexpr (std::is_same_v<Operation, AddOp>)
    {
      return Operation::compute(input, other);
    }
    else if constexpr (std::is_same_v<Operation, SubOp>)
    {
      return Operation::compute(input, other);
    }
    else if constexpr (std::is_same_v<Operation, MulOp>)
    {
      return Operation::compute(input, other);
    }
    else if constexpr (std::is_same_v<Operation, DivOp>)
    {
      return Operation::compute(input, other);
    }
    else
    {
      static_assert(false, "Unsupported operation");
    }
  }

  template <class _T, int _WG, int BIG_TILE_SIZE_N, int BIG_TILE_SIZE_K, int M_SWIZZLE, typename Operation, bool order_flag, class _T0, class _T1>
  __global__ void operator_tn_big_tile_kernel(const void *__restrict a, const void *__restrict b, void *__restrict c,
                                              const int N, const int K, int stride0, int stride2, bool types_match)
  {
    // pad LDS row by dword
    constexpr uint32_t LDS_PAD = 4 / sizeof(_T);
    constexpr uint32_t element_size = sizeof(_T); // in bytes
    constexpr uint32_t elements_in_16B = 16 / element_size;

    union BLOCK_16B
    {
      _T e[elements_in_16B];
      __uint128_t ow;
    };

    // Round up processing to next full tile
    const uint32_t n_tiles = (N + BIG_TILE_SIZE_N - 1) / BIG_TILE_SIZE_N;
    const uint32_t k_tiles = (K + BIG_TILE_SIZE_K - 1) / BIG_TILE_SIZE_K;
    const uint32_t nk_tiles = n_tiles * k_tiles;
    const uint32_t m_tiles = gridDim.x / nk_tiles;
    const uint32_t m_tile_swizzle = blockIdx.x / nk_tiles / M_SWIZZLE * M_SWIZZLE;
    /// do m_swizzle when there are enough m_tiles
    const bool swizzle_m = m_tile_swizzle + M_SWIZZLE <= m_tiles;
    const uint32_t current_m = swizzle_m ? m_tile_swizzle + blockIdx.x % M_SWIZZLE : blockIdx.x / nk_tiles;

    const uint64_t stride_k = N;
    const uint64_t out_stride_nk = N * K;

    const uint32_t current_nk = swizzle_m ? blockIdx.x / M_SWIZZLE % nk_tiles : blockIdx.x % nk_tiles;
    const uint32_t ti = current_nk / k_tiles;
    const uint32_t tj = current_nk % k_tiles;

    __shared__ _T0 sa[BIG_TILE_SIZE_N][BIG_TILE_SIZE_K + LDS_PAD];

    const uint32_t current_n_size = (ti == (n_tiles - 1) && (N % BIG_TILE_SIZE_N) != 0) ? (N % BIG_TILE_SIZE_N) : BIG_TILE_SIZE_N;
    const uint32_t current_k_size = (tj == (k_tiles - 1) && (K % BIG_TILE_SIZE_K) != 0) ? (K % BIG_TILE_SIZE_K) : BIG_TILE_SIZE_K;
    // use 128bit load&store whenever possible
    if (current_n_size % elements_in_16B == 0 && current_k_size % 8 == 0)
    {
      // Copy full tile with large loads
      constexpr uint32_t row_bytes = BIG_TILE_SIZE_K;
      constexpr uint32_t ld_per_row = row_bytes / elements_in_16B;
      constexpr uint32_t rows_per_wg = _WG / ld_per_row;
      constexpr uint32_t vmem_per_thread = BIG_TILE_SIZE_N / rows_per_wg;
      // Make sure WG isn't too large
      static_assert(vmem_per_thread >= 1);

      const _T0 *pat = (const _T0 *)a + tj * row_bytes + ti * BIG_TILE_SIZE_N * stride2 + current_m * stride0;
#pragma unroll
      for (uint32_t t = 0; t < vmem_per_thread; t++)
      {
        uint32_t col = threadIdx.x % ld_per_row;
        uint32_t row = threadIdx.x / ld_per_row + t * rows_per_wg;
        uint64_t offset = (col * elements_in_16B < current_k_size && row < current_n_size) ? row * stride2 + col * elements_in_16B : 0;
        const _T0 *pfa = (const _T0 *)(pat + offset);
        // BLOCK_16B d;
        // d.ow = *pfa;
#pragma unroll
        for (uint32_t i = 0; i < elements_in_16B; i++)
        {
          sa[row][col * elements_in_16B + i] = pfa[i];
        }
      }
      __syncthreads();
      // Copy full tile with large loads
      constexpr uint32_t row_bytes_wr = BIG_TILE_SIZE_N;
      constexpr uint32_t vmem_per_row_wr = row_bytes_wr / elements_in_16B;
      constexpr uint32_t rows_per_wg_wr = _WG / vmem_per_row_wr;
      constexpr uint32_t wr_per_row = BIG_TILE_SIZE_K / rows_per_wg_wr;
      // Make sure WG isn't too large
      static_assert(wr_per_row >= 1);

      const _T1 *pb = (const _T1 *)b + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
      _T *pc = (_T *)c + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
#pragma unroll
      for (uint32_t t = 0; t < vmem_per_thread; t++)
      {
        uint32_t col = threadIdx.x % vmem_per_row_wr;
        uint32_t row = threadIdx.x / vmem_per_row_wr + t * rows_per_wg_wr;
        if (col * elements_in_16B < current_n_size && row < current_k_size)
        {
          uint64_t offset = row * stride_k + col * elements_in_16B;
          BLOCK_16B d;
          if (types_match)
          {
            const __uint128_t *pfb = (const __uint128_t *)(pb + offset);
            d.ow = *pfb;
// Transpose tile on read from LDS
#pragma unroll
            for (uint32_t i = 0; i < elements_in_16B; i++)
            {
              d.e[i] = performOperation<_T, Operation, order_flag>(static_cast<_T>(sa[col * elements_in_16B + i][row]), d.e[i]);
            }
            __uint128_t *pfc = (__uint128_t *)(pc + offset);
            *pfc = d.ow;
          }
          else
          {
            const _T1 *pfb = (const _T1 *)(pb + offset);
#pragma unroll
            for (uint32_t i = 0; i < elements_in_16B; i++)
            {
              float a = static_cast<float>(sa[col * elements_in_16B + i][row]);
              float b = static_cast<float>(pfb[i]);
              float c = performOperation<float, Operation, order_flag>(a, b);
              d.e[i] = static_cast<_T>(c);
            }
            __uint128_t *pfc = (__uint128_t *)(pc + offset);
            *pfc = d.ow;
          }
        }
      }
    }
    else
    {
      // Copy partial tiles with element accesses
      constexpr uint32_t row_bytes = BIG_TILE_SIZE_K;
      constexpr uint32_t ld_per_row = BIG_TILE_SIZE_K;
      constexpr uint32_t rows_per_wg = _WG / ld_per_row;
      constexpr uint32_t vmem_per_thread = BIG_TILE_SIZE_N / rows_per_wg;
      // Make sure WG isn't too large
      static_assert(vmem_per_thread >= 1);

      const _T0 *pat = (const _T0 *)a + ti * BIG_TILE_SIZE_N * stride2 + tj * row_bytes + current_m * stride0;
#pragma unroll
      for (uint32_t t = 0; t < vmem_per_thread; t++)
      {
        uint32_t col = threadIdx.x % ld_per_row;
        uint32_t row = threadIdx.x / ld_per_row + t * rows_per_wg;
        uint64_t offset = (col < current_k_size && row < current_n_size) ? row * stride2 + col : 0;
        const _T0 *pfa = (const _T0 *)(pat + offset);
        sa[row][col] = *pfa;
      }
      __syncthreads();

      // Copy full tile with large loads
      constexpr uint32_t row_bytes_wr = BIG_TILE_SIZE_N;
      constexpr uint32_t vmem_per_row_wr = BIG_TILE_SIZE_N;
      constexpr uint32_t rows_per_wg_wr = _WG / vmem_per_row_wr;
      constexpr uint32_t wr_per_row = BIG_TILE_SIZE_K / rows_per_wg_wr;
      const _T1 *pb = (const _T1 *)b + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
      _T *pc = (_T *)c + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
#pragma unroll
      for (uint32_t t = 0; t < wr_per_row; t++)
      {
        uint32_t col = threadIdx.x % vmem_per_row_wr;
        uint32_t row = threadIdx.x / vmem_per_row_wr + t * rows_per_wg_wr;
        if (col < current_n_size && row < current_k_size)
        {
          uint64_t offset = row * stride_k + col;
          const _T1 *pfb = (const _T1 *)(pb + offset);
          _T *pfc = (_T *)(pc + offset);
          if (types_match)
          {
            *pfc = performOperation<_T, Operation, order_flag>(static_cast<_T>(sa[col][row]), static_cast<_T>(*pfb));
          }
          else
          {
            float a = static_cast<float>(sa[col][row]);
            float b = static_cast<float>(*pfb);
            float c = performOperation<float, Operation, order_flag>(a, b);
            *pfc = static_cast<_T>(c);
          }
        }
      }
    }
  }

  template <class _T, int _WG, int BIG_TILE_SIZE_N, int BIG_TILE_SIZE_K, int M_SWIZZLE, typename Operation, bool order_flag, class _T0, class _T1>
  __global__ void operator_bcast_big_tile_kernel(const void *__restrict a, const void *__restrict b, void *__restrict c,
                                                 const int N, const int K, bool types_match)
  {
    constexpr uint32_t element_size = sizeof(_T); // in bytes
    constexpr uint32_t elements_in_16B = 16 / element_size;

    union BLOCK_16B
    {
      _T e[elements_in_16B];
      __uint128_t ow;
    };

    // Round up processing to next full tile
    const uint32_t n_tiles = (N + BIG_TILE_SIZE_N - 1) / BIG_TILE_SIZE_N;
    const uint32_t k_tiles = (K + BIG_TILE_SIZE_K - 1) / BIG_TILE_SIZE_K;
    const uint32_t nk_tiles = n_tiles * k_tiles;
    const uint32_t m_tiles = gridDim.x / nk_tiles;
    const uint32_t m_tile_swizzle = blockIdx.x / nk_tiles / M_SWIZZLE * M_SWIZZLE;
    /// do m_swizzle when there are enough m_tiles
    const bool swizzle_m = m_tile_swizzle + M_SWIZZLE <= m_tiles;
    const uint32_t current_m = swizzle_m ? m_tile_swizzle + blockIdx.x % M_SWIZZLE : blockIdx.x / nk_tiles;

    const uint64_t stride_k = N;
    const uint64_t out_stride_nk = N * K;

    const uint32_t current_nk = swizzle_m ? blockIdx.x / M_SWIZZLE % nk_tiles : blockIdx.x % nk_tiles;
    const uint32_t ti = current_nk / k_tiles;
    const uint32_t tj = current_nk % k_tiles;

    const uint32_t current_n_size = (ti == (n_tiles - 1) && (N % BIG_TILE_SIZE_N) != 0) ? (N % BIG_TILE_SIZE_N) : BIG_TILE_SIZE_N;
    const uint32_t current_k_size = (tj == (k_tiles - 1) && (K % BIG_TILE_SIZE_K) != 0) ? (K % BIG_TILE_SIZE_K) : BIG_TILE_SIZE_K;

    // use 128bit load&store whenever possible
    if (current_n_size % 8 == 0 && current_k_size % elements_in_16B == 0)
    {
      // Copy full tile with large loads
      constexpr uint32_t row_bytes_wr = BIG_TILE_SIZE_N;
      constexpr uint32_t vmem_per_row_wr = row_bytes_wr / elements_in_16B;
      constexpr uint32_t rows_per_wg_wr = _WG / vmem_per_row_wr;
      constexpr uint32_t wr_per_row = BIG_TILE_SIZE_K / rows_per_wg_wr;
      // Make sure WG isn't too large
      static_assert(wr_per_row >= 1);

      const _T0 *pa = (const _T0 *)a + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
      const _T1 *pb = (const _T1 *)b + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr;
      const _T *pc = (const _T *)c + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
#pragma unroll
      for (uint32_t t = 0; t < wr_per_row; t++)
      {
        uint32_t col = threadIdx.x % vmem_per_row_wr;
        uint32_t row = threadIdx.x / vmem_per_row_wr + t * rows_per_wg_wr;
        if (col * elements_in_16B < current_n_size && row < current_k_size)
        {
          BLOCK_16B d, f;
          uint64_t offset = row * stride_k + col * elements_in_16B;
          if (types_match)
          {
            const __uint128_t *pfa = (const __uint128_t *)(pa + offset);
            const __uint128_t *pfb = (const __uint128_t *)(pb + offset);
            f.ow = *pfa;
            d.ow = *pfb;
#pragma unroll
            for (uint32_t i = 0; i < elements_in_16B; i++)
            {
              d.e[i] = performOperation<_T, Operation, order_flag>(static_cast<_T>(f.e[i]), static_cast<_T>(d.e[i]));
            }
            __uint128_t *pfc = (__uint128_t *)(pc + offset);
            *pfc = d.ow;
          }
          else
          {
            const _T0 *pfa = (const _T0 *)(pa + offset);
            const _T1 *pfb = (const _T1 *)(pb + offset);
#pragma unroll
            for (uint32_t i = 0; i < elements_in_16B; i++)
            {
              float a = static_cast<float>(pfa[i]);
              float b = static_cast<float>(pfb[i]);
              float c = performOperation<float, Operation, order_flag>(a, b);
              d.e[i] = static_cast<_T>(c);
            }
            __uint128_t *pfc = (__uint128_t *)(pc + offset);
            *pfc = d.ow;
          }
        }
      }
    }
    else
    {
      // Copy full tile with large loads
      constexpr uint32_t row_bytes_wr = BIG_TILE_SIZE_N;
      constexpr uint32_t vmem_per_row_wr = BIG_TILE_SIZE_N;
      constexpr uint32_t rows_per_wg_wr = _WG / vmem_per_row_wr;
      constexpr uint32_t wr_per_row = BIG_TILE_SIZE_K / rows_per_wg_wr;
      const _T0 *pa = (const _T0 *)a + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
      const _T1 *pb = (const _T1 *)b + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr;
      const _T *pc = (const _T *)c + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
#pragma unroll
      for (uint32_t t = 0; t < wr_per_row; t++)
      {
        uint32_t col = threadIdx.x % vmem_per_row_wr;
        uint32_t row = threadIdx.x / vmem_per_row_wr + t * rows_per_wg_wr;
        if (col < current_n_size && row < current_k_size)
        {
          uint64_t offset = row * stride_k + col;
          const _T0 *pfa = (const _T0 *)(pa + offset);
          const _T1 *pfb = (const _T1 *)(pb + offset);
          _T *pfc = (_T *)(pc + offset);
          if (types_match)
          {
            *pfc = performOperation<_T, Operation, order_flag>(static_cast<_T>(*pfa), static_cast<_T>(*pfb));
          }
          else
          {
            float a = static_cast<float>(*pfa);
            float b = static_cast<float>(*pfb);
            float c = performOperation<float, Operation, order_flag>(a, b);
            *pfc = static_cast<_T>(c);
          }
        }
      }
    }
  }

  template <class _T, int _WG, int BIG_TILE_SIZE_N, int BIG_TILE_SIZE_K, int M_SWIZZLE, typename Operation, bool order_flag, class _T0, class _T1>
  __global__ void operator_bcast1_big_tile_kernel(const void *__restrict a, const void *__restrict b, void *__restrict c,
                                                  const int N, const int K, bool types_match)
  {
    // pad LDS row by dword
    constexpr uint32_t element_size = sizeof(_T); // in bytes
    constexpr uint32_t elements_in_16B = 16 / element_size;

    union BLOCK_16B
    {
      _T e[elements_in_16B];
      __uint128_t ow;
    };

    // Round up processing to next full tile
    const uint32_t n_tiles = (N + BIG_TILE_SIZE_N - 1) / BIG_TILE_SIZE_N;
    const uint32_t k_tiles = (K + BIG_TILE_SIZE_K - 1) / BIG_TILE_SIZE_K;
    const uint32_t nk_tiles = n_tiles * k_tiles;
    const uint32_t m_tiles = gridDim.x / nk_tiles;
    const uint32_t m_tile_swizzle = blockIdx.x / nk_tiles / M_SWIZZLE * M_SWIZZLE;
    /// do m_swizzle when there are enough m_tiles
    const bool swizzle_m = m_tile_swizzle + M_SWIZZLE <= m_tiles;
    const uint32_t current_m = swizzle_m ? m_tile_swizzle + blockIdx.x % M_SWIZZLE : blockIdx.x / nk_tiles;

    const uint64_t stride_k = N;
    const uint64_t out_stride_nk = N * K;

    const uint32_t current_nk = swizzle_m ? blockIdx.x / M_SWIZZLE % nk_tiles : blockIdx.x % nk_tiles;
    const uint32_t ti = current_nk / k_tiles;
    const uint32_t tj = current_nk % k_tiles;

    const uint32_t current_n_size = (ti == (n_tiles - 1) && (N % BIG_TILE_SIZE_N) != 0) ? (N % BIG_TILE_SIZE_N) : BIG_TILE_SIZE_N;
    const uint32_t current_k_size = (tj == (k_tiles - 1) && (K % BIG_TILE_SIZE_K) != 0) ? (K % BIG_TILE_SIZE_K) : BIG_TILE_SIZE_K;

    // use 128bit load&store whenever possible
    if (current_n_size % 8 == 0 && current_k_size % elements_in_16B == 0)
    {
      // Copy full tile with large loads
      constexpr uint32_t row_bytes_wr = BIG_TILE_SIZE_N;
      constexpr uint32_t vmem_per_row_wr = row_bytes_wr / elements_in_16B;
      constexpr uint32_t rows_per_wg_wr = _WG / vmem_per_row_wr;
      constexpr uint32_t wr_per_row = BIG_TILE_SIZE_K / rows_per_wg_wr;
      // Make sure WG isn't too large
      static_assert(wr_per_row >= 1);

      const _T0 *pa = (const _T0 *)a + ti * row_bytes_wr + current_m * stride_k;
      const _T1 *pb = (const _T1 *)b + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
      const _T *pc = (const _T *)c + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
#pragma unroll
      for (uint32_t t = 0; t < wr_per_row; t++)
      {
        uint32_t col = threadIdx.x % vmem_per_row_wr;
        uint32_t row = threadIdx.x / vmem_per_row_wr + t * rows_per_wg_wr;
        if (col * elements_in_16B < current_n_size && row < current_k_size)
        {
          uint64_t offset_a = col * elements_in_16B;
          uint64_t offset = row * stride_k + col * elements_in_16B;
          BLOCK_16B d, f;
          if (types_match)
          {
            const __uint128_t *pfa = (const __uint128_t *)(pa + offset_a);
            const __uint128_t *pfb = (const __uint128_t *)(pb + offset);
            f.ow = *pfa;
            d.ow = *pfb;
#pragma unroll
            for (uint32_t i = 0; i < elements_in_16B; i++)
            {
              d.e[i] = performOperation<_T, Operation, order_flag>(static_cast<_T>(f.e[i]), static_cast<_T>(d.e[i]));
            }
            __uint128_t *pfc = (__uint128_t *)(pc + offset);
            *pfc = d.ow;
          }
          else
          {
            const _T0 *pfa = (const _T0 *)(pa + offset_a);
            const _T1 *pfb = (const _T1 *)(pb + offset);
#pragma unroll
            for (uint32_t i = 0; i < elements_in_16B; i++)
            {
              float a = static_cast<float>(pfa[i]);
              float b = static_cast<float>(pfb[i]);
              float c = performOperation<float, Operation, order_flag>(a, b);
              d.e[i] = static_cast<_T>(c);
            }
            __uint128_t *pfc = (__uint128_t *)(pc + offset);
            *pfc = d.ow;
          }
        }
      }
    }
    else
    {
      // Copy full tile with large loads
      constexpr uint32_t row_bytes_wr = BIG_TILE_SIZE_N;
      constexpr uint32_t vmem_per_row_wr = BIG_TILE_SIZE_N;
      constexpr uint32_t rows_per_wg_wr = _WG / vmem_per_row_wr;
      constexpr uint32_t wr_per_row = BIG_TILE_SIZE_K / rows_per_wg_wr;
      const _T0 *pa = (const _T0 *)a + ti * row_bytes_wr + current_m * stride_k;
      const _T1 *pb = (const _T1 *)b + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
      const _T *pc = (const _T *)c + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
#pragma unroll
      for (uint32_t t = 0; t < wr_per_row; t++)
      {
        uint32_t col = threadIdx.x % vmem_per_row_wr;
        uint32_t row = threadIdx.x / vmem_per_row_wr + t * rows_per_wg_wr;
        if (col < current_n_size && row < current_k_size)
        {
          uint64_t offset_a = col;
          uint64_t offset = row * stride_k + col;
          const _T0 *pfa = (const _T0 *)(pa + offset_a);
          const _T1 *pfb = (const _T1 *)(pb + offset);
          _T *pfc = (_T *)(pc + offset);
          if (types_match)
          {
            *pfc = performOperation<_T, Operation, order_flag>(static_cast<_T>(*pfa), static_cast<_T>(*pfb));
          }
          else
          {
            float a = static_cast<float>(*pfa);
            float b = static_cast<float>(*pfb);
            float c = performOperation<float, Operation, order_flag>(a, b);
            *pfc = static_cast<_T>(c);
          }
        }
      }
    }
  }

  template <class _T, int _rows, typename Operation, bool order_flag, class _T0, class _T1>
  __global__ void operator_bcast_tile_kernel(const void *__restrict a, const void *__restrict b, void *__restrict c,
                                             const int M, const int N, const int K, bool types_match)
  {
    constexpr uint32_t element_size = sizeof(_T); // in bytes
    constexpr uint32_t elements_in_16B = 16 / element_size;
    uint64_t idx = (uint64_t)blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t n_tiles = N / _rows;
    uint32_t k_tiles = K / elements_in_16B;
    if (idx < (uint64_t)M * n_tiles * k_tiles)
    {
      uint32_t ti = idx / (k_tiles * n_tiles);
      uint64_t idx_block = idx % (k_tiles * n_tiles);
      uint32_t tj = (idx_block / k_tiles) % n_tiles;
      uint32_t tk = idx_block % k_tiles;
      for (int row = 0; row < _rows; row++)
      {
        uint64_t offset_b = (uint64_t)(tj + row * n_tiles) * K + tk * elements_in_16B;
        uint64_t offset_ac = (uint64_t)(tj + row * n_tiles) * K + tk * elements_in_16B + (uint64_t)ti * N * K;
        const _T0 *pa = reinterpret_cast<const _T0 *>(a) + offset_ac;
        const _T1 *pb = reinterpret_cast<const _T1 *>(b) + offset_b;
        _T *pc = reinterpret_cast<_T *>(c) + offset_ac;
        for (int col = 0; col < elements_in_16B; col++)
        {
          const _T0 *pfa = pa + col;
          const _T1 *pfb = pb + col;
          _T *pfc = pc + col;
          if (types_match)
          {
            *pfc = performOperation<_T, Operation, order_flag>(static_cast<_T>(*pfa), static_cast<_T>(*pfb));
          }
          else
          {
            float t0 = static_cast<float>(*pfa);
            float t1 = static_cast<float>(*pfb);
            float t2 = performOperation<float, Operation, order_flag>(t0, t1);
            *pfc = static_cast<_T>(t2);
          }
        }
      }
    }
  }

  template <class _T, int _rows, typename Operation, bool order_flag, class _T0, class _T1>
  __global__ void operator_contiguous_kernel(const void *__restrict a, const void *__restrict b, void *__restrict c,
                                             const int M, const int N, const int K, bool types_match)
  {
    constexpr uint32_t element_size = sizeof(_T); // in bytes
    constexpr uint32_t elements_in_16B = 16 / element_size;
    uint64_t idx = (uint64_t)blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t n_tiles = N / _rows;
    uint32_t k_tiles = K / elements_in_16B;
    if (idx < (uint64_t)M * n_tiles * k_tiles)
    {
      uint32_t ti = idx / (k_tiles * n_tiles);
      uint64_t idx_block = idx % (k_tiles * n_tiles);
      uint32_t tj = (idx_block / k_tiles) % n_tiles;
      uint32_t tk = idx_block % k_tiles;
      for (int row = 0; row < _rows; row++)
      {
        uint64_t offset = (uint64_t)(tj + row * n_tiles) * K + tk * elements_in_16B + (uint64_t)ti * N * K;
        const _T0 *pa = reinterpret_cast<const _T0 *>(a) + offset;
        const _T1 *pb = reinterpret_cast<const _T1 *>(b) + offset;
        _T *pc = reinterpret_cast<_T *>(c) + offset;
        for (int col = 0; col < elements_in_16B; col++)
        {
          const _T0 *pfa = pa + col;
          const _T1 *pfb = pb + col;
          _T *pfc = pc + col;
          if (types_match)
          {
            *pfc = performOperation<_T, Operation, order_flag>(static_cast<_T>(*pfa), static_cast<_T>(*pfb));
          }
          else
          {
            float t0 = static_cast<float>(*pfa);
            float t1 = static_cast<float>(*pfb);
            float t2 = performOperation<float, Operation, order_flag>(t0, t1);
            *pfc = static_cast<_T>(t2);
          }
        }
      }
    }
  }

  template <class _T, typename Operation, class _T0, class _T1>
  __global__ void operator_element_kernel(const void *__restrict a, const void *__restrict b, void *__restrict c,
                                          const int size, bool types_match)
  {
    constexpr uint32_t element_size = sizeof(_T); // in bytes
    constexpr uint32_t elements_in_16B = 16 / element_size;
    uint64_t idx = ((uint64_t)blockIdx.x * blockDim.x + threadIdx.x);
    if (idx * elements_in_16B < size)
    {
      int offset = idx * elements_in_16B;
      const _T0 *pa = reinterpret_cast<const _T0 *>(a) + offset;
      const _T1 *pb = reinterpret_cast<const _T1 *>(b) + offset;
      _T *pc = reinterpret_cast<_T *>(c) + offset;
#pragma unroll
      for (uint32_t v = 0; v < elements_in_16B; v++)
      {
        if (types_match)
        {
          pc[v] = performOperation<_T, Operation, true>(static_cast<_T>(pa[v]), static_cast<_T>(pb[v]));
        }
        else
        {
          float t0 = static_cast<float>(pa[v]);
          float t1 = static_cast<float>(pb[v]);
          float t2 = performOperation<float, Operation, true>(t0, t1);
          pc[v] = static_cast<_T>(t2);
        }
      }
    }
  }

  template <class _T, int _WG, int BIG_TILE_SIZE_N, int BIG_TILE_SIZE_K, int M_SWIZZLE, typename Operation, bool order_flag, class _T0, class _T1>
  __global__ void operator_contiguous_big_tile_kernel(const void *__restrict a, const void *__restrict b, void *__restrict c,
                                                      const int N, const int K, bool types_match)
  {
    constexpr uint32_t element_size = sizeof(_T); // in bytes
    constexpr uint32_t elements_in_16B = 16 / element_size;

    union BLOCK_16B
    {
      _T e[elements_in_16B];
      __uint128_t ow;
    };

    // Round up processing to next full tile
    const uint32_t n_tiles = (N + BIG_TILE_SIZE_N - 1) / BIG_TILE_SIZE_N;
    const uint32_t k_tiles = (K + BIG_TILE_SIZE_K - 1) / BIG_TILE_SIZE_K;
    const uint32_t nk_tiles = n_tiles * k_tiles;
    const uint32_t m_tiles = gridDim.x / nk_tiles;
    const uint32_t m_tile_swizzle = blockIdx.x / nk_tiles / M_SWIZZLE * M_SWIZZLE;
    /// do m_swizzle when there are enough m_tiles
    const bool swizzle_m = m_tile_swizzle + M_SWIZZLE <= m_tiles;
    const uint32_t current_m = swizzle_m ? m_tile_swizzle + blockIdx.x % M_SWIZZLE : blockIdx.x / nk_tiles;

    const uint64_t stride_k = N;
    const uint64_t out_stride_nk = N * K;

    const uint32_t current_nk = swizzle_m ? blockIdx.x / M_SWIZZLE % nk_tiles : blockIdx.x % nk_tiles;
    const uint32_t ti = current_nk / k_tiles;
    const uint32_t tj = current_nk % k_tiles;

    const uint32_t current_n_size = (ti == (n_tiles - 1) && (N % BIG_TILE_SIZE_N) != 0) ? (N % BIG_TILE_SIZE_N) : BIG_TILE_SIZE_N;
    const uint32_t current_k_size = (tj == (k_tiles - 1) && (K % BIG_TILE_SIZE_K) != 0) ? (K % BIG_TILE_SIZE_K) : BIG_TILE_SIZE_K;

    // use 128bit load&store whenever possible
    if (current_n_size % 8 == 0 && current_k_size % elements_in_16B == 0)
    {
      // Copy full tile with large loads
      constexpr uint32_t row_bytes_wr = BIG_TILE_SIZE_N;
      constexpr uint32_t vmem_per_row_wr = row_bytes_wr / elements_in_16B;
      constexpr uint32_t rows_per_wg_wr = _WG / vmem_per_row_wr;
      constexpr uint32_t wr_per_row = BIG_TILE_SIZE_K / rows_per_wg_wr;
      // Make sure WG isn't too large
      static_assert(wr_per_row >= 1);

      const _T0 *pa = (const _T0 *)a + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
      const _T1 *pb = (const _T1 *)b + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
      const _T *pc = (const _T *)c + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
#pragma unroll
      for (uint32_t t = 0; t < wr_per_row; t++)
      {
        uint32_t col = threadIdx.x % vmem_per_row_wr;
        uint32_t row = threadIdx.x / vmem_per_row_wr + t * rows_per_wg_wr;
        if (col * elements_in_16B < current_n_size && row < current_k_size)
        {
          BLOCK_16B d, f;
          uint64_t offset = row * stride_k + col * elements_in_16B;
          if (types_match)
          {
            const __uint128_t *pfa = (const __uint128_t *)(pa + offset);
            const __uint128_t *pfb = (const __uint128_t *)(pb + offset);
            f.ow = *pfa;
            d.ow = *pfb;
#pragma unroll
            for (uint32_t i = 0; i < elements_in_16B; i++)
            {
              d.e[i] = performOperation<_T, Operation, order_flag>(static_cast<_T>(f.e[i]), static_cast<_T>(d.e[i]));
            }
            __uint128_t *pfc = (__uint128_t *)(pc + offset);
            *pfc = d.ow;
          }
          else
          {
            const _T0 *pfa = (const _T0 *)(pa + offset);
            const _T1 *pfb = (const _T1 *)(pb + offset);
#pragma unroll
            for (uint32_t i = 0; i < elements_in_16B; i++)
            {
              float a = static_cast<float>(pfa[i]);
              float b = static_cast<float>(pfb[i]);
              float c = performOperation<float, Operation, order_flag>(a, b);
              d.e[i] = static_cast<_T>(c);
            }
            __uint128_t *pfc = (__uint128_t *)(pc + offset);
            *pfc = d.ow;
          }
        }
      }
    }
    else
    {
      // Copy full tile with large loads
      constexpr uint32_t row_bytes_wr = BIG_TILE_SIZE_N;
      constexpr uint32_t vmem_per_row_wr = BIG_TILE_SIZE_N;
      constexpr uint32_t rows_per_wg_wr = _WG / vmem_per_row_wr;
      constexpr uint32_t wr_per_row = BIG_TILE_SIZE_K / rows_per_wg_wr;
      const _T0 *pa = (const _T0 *)a + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
      const _T1 *pb = (const _T1 *)b + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
      const _T *pc = (const _T *)c + tj * BIG_TILE_SIZE_K * stride_k + ti * row_bytes_wr + current_m * out_stride_nk;
#pragma unroll
      for (uint32_t t = 0; t < wr_per_row; t++)
      {
        uint32_t col = threadIdx.x % vmem_per_row_wr;
        uint32_t row = threadIdx.x / vmem_per_row_wr + t * rows_per_wg_wr;
        if (col < current_n_size && row < current_k_size)
        {
          uint64_t offset = row * stride_k + col;
          const _T0 *pfa = (const _T0 *)(pa + offset);
          const _T1 *pfb = (const _T1 *)(pb + offset);
          _T *pfc = (_T *)(pc + offset);
          if (types_match)
          {
            *pfc = performOperation<_T, Operation, order_flag>(static_cast<_T>(*pfa), static_cast<_T>(*pfb));
          }
          else
          {
            float a = static_cast<float>(*pfa);
            float b = static_cast<float>(*pfb);
            float c = performOperation<float, Operation, order_flag>(a, b);
            *pfc = static_cast<_T>(c);
          }
        }
      }
    }
  }
} // namespace aiter

std::vector<int64_t> broadcastShapes(const torch::Tensor &tensor1, const torch::Tensor &tensor2)
{
  auto shape1 = tensor1.sizes().vec();
  auto shape2 = tensor2.sizes().vec();

  int64_t max_dim = std::max(shape1.size(), shape2.size());
  shape1.insert(shape1.begin(), max_dim - shape1.size(), 1);
  shape2.insert(shape2.begin(), max_dim - shape2.size(), 1);

  std::vector<int64_t> result_shape(max_dim, 1);
  for (int64_t i = 0; i < max_dim; ++i)
  {
    if (shape1[i] == 1)
    {
      result_shape[i] = shape2[i];
    }
    else if (shape2[i] == 1)
    {
      result_shape[i] = shape1[i];
    }
    else if (shape1[i] == shape2[i])
    {
      result_shape[i] = shape1[i];
    }
    else
    {
      throw std::invalid_argument("Incompatible shapes for binary operator.");
    }
  }

  return result_shape;
}

template <int pattern, typename Operation, class _T0, class _T1>
struct BinaryOperationPattern;

// PATTERN_TRANSPOSE
template <typename Operation, class _T0, class _T1>
struct BinaryOperationPattern<1, Operation, _T0, _T1>
{
  static void apply(torch::Tensor &input, torch::Tensor &other, torch::Tensor &output, bool order_flag)
  {
    int dim = input.dim();
    auto shape = output.sizes().vec();
    void *buf_a = reinterpret_cast<void *>(input.data_ptr());
    void *buf_b = reinterpret_cast<void *>(other.data_ptr());
    void *buf_c = reinterpret_cast<void *>(output.data_ptr());

    int num_elements = output.numel();
    int rem_dim_size = num_elements / (shape[dim - 2] * shape[dim - 1]);
    int M = dim == 2 ? 1 : rem_dim_size;
    int N = shape[dim - 2];
    int K = shape[dim - 1];

    auto tensor_not_conti = input.is_contiguous() ? other : input;
    int stride0 = tensor_not_conti.stride(0);
    int stride2 = tensor_not_conti.stride(2);
    constexpr uint32_t BIG_TILE_SIZE_N = 64;
    constexpr uint32_t BIG_TILE_SIZE_K = 64;
    constexpr uint32_t M_SWIZZLE = 8;
    const int grid_x = M * ((N + BIG_TILE_SIZE_N - 1) / BIG_TILE_SIZE_N) * ((K + BIG_TILE_SIZE_K - 1) / BIG_TILE_SIZE_K);
    const dim3 grid_dim(grid_x, 1, 1);
    const dim3 block_dim(256, 1, 1);
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    bool types_match = typeid(_T0) == typeid(_T1);

    if (order_flag)
    {
      VLLM_DISPATCH_FLOATING_TYPES(
          output.scalar_type(), "operator_tn_big_tile_kernel", [&]
          { aiter::operator_tn_big_tile_kernel<scalar_t, 256, BIG_TILE_SIZE_N, BIG_TILE_SIZE_K, M_SWIZZLE, Operation, true, _T0, _T1>
                <<<grid_dim, block_dim, 0, stream>>>(buf_a, buf_b, buf_c, K, N, stride0, stride2, types_match); });
    }
    else
    {
      VLLM_DISPATCH_FLOATING_TYPES(
          output.scalar_type(), "operator_tn_big_tile_kernel", [&]
          { aiter::operator_tn_big_tile_kernel<scalar_t, 256, BIG_TILE_SIZE_N, BIG_TILE_SIZE_K, M_SWIZZLE, Operation, false, _T1, _T0>
                <<<grid_dim, block_dim, 0, stream>>>(buf_b, buf_a, buf_c, K, N, stride0, stride2, types_match); });
    }
  }
};

// PATTERN_BROADCAST_0
template <typename Operation, class _T0, class _T1>
struct BinaryOperationPattern<2, Operation, _T0, _T1>
{
  static void apply(torch::Tensor &input, torch::Tensor &other, torch::Tensor &output, bool order_flag)
  {
    int dim = input.dim();
    auto shape = output.sizes().vec();

    void *buf_a = reinterpret_cast<void *>(input.data_ptr());
    void *buf_b = reinterpret_cast<void *>(other.data_ptr());
    void *buf_c = reinterpret_cast<void *>(output.data_ptr());
    int num_elements = output.numel();
    int rem_dim_size = num_elements / (shape[dim - 2] * shape[dim - 1]);
    int M = dim != 3 ? 1 : rem_dim_size;
    int N = shape[dim - 2];
    int K = shape[dim - 1];
    if (dim == 4)
    {
      N = shape[0] * shape[1] * shape[2];
      K = shape[3];
    }
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    bool types_match = typeid(_T0) == typeid(_T1);

    const uint32_t rows = 8;
    int vec = 16 / output.element_size();
    if (N % rows == 0 && K % vec == 0)
    {
      constexpr uint32_t wg = 64;
      int grid_x = (num_elements / (rows * vec) + wg - 1) / wg;
      const dim3 grid_dim(grid_x, 1, 1);
      const dim3 block_dim(wg, 1, 1);

      if (order_flag)
      {
        VLLM_DISPATCH_FLOATING_TYPES(
            output.scalar_type(), "operator_bcast_tile_kernel", [&]
            { aiter::operator_bcast_tile_kernel<scalar_t, rows, Operation, true, _T0, _T1>
                  <<<grid_dim, block_dim, 0, stream>>>(buf_a, buf_b, buf_c, M, N, K, types_match); });
      }
      else
      {
        VLLM_DISPATCH_FLOATING_TYPES(
            output.scalar_type(), "operator_bcast_tile_kernel", [&]
            { aiter::operator_bcast_tile_kernel<scalar_t, rows, Operation, false, _T1, _T0>
                  <<<grid_dim, block_dim, 0, stream>>>(buf_b, buf_a, buf_c, M, N, K, types_match); });
      }
    }
    else
    {
      constexpr uint32_t BIG_TILE_SIZE_N = 64;
      constexpr uint32_t BIG_TILE_SIZE_K = 64;
      constexpr uint32_t M_SWIZZLE = 8;
      const int grid_x = M * ((N + BIG_TILE_SIZE_N - 1) / BIG_TILE_SIZE_N) * ((K + BIG_TILE_SIZE_K - 1) / BIG_TILE_SIZE_K);
      const dim3 grid_dim(grid_x, 1, 1);
      const dim3 block_dim(256, 1, 1);

      if (order_flag)
      {
        VLLM_DISPATCH_FLOATING_TYPES(
            output.scalar_type(), "operator_bcast_big_tile_kernel", [&]
            { aiter::operator_bcast_big_tile_kernel<scalar_t, 256, BIG_TILE_SIZE_N, BIG_TILE_SIZE_K, M_SWIZZLE, Operation, true, _T0, _T1>
                  <<<grid_dim, block_dim, 0, stream>>>(buf_a, buf_b, buf_c, K, N, types_match); });
      }
      else
      {
        VLLM_DISPATCH_FLOATING_TYPES(
            output.scalar_type(), "operator_bcast_big_tile_kernel", [&]
            { aiter::operator_bcast_big_tile_kernel<scalar_t, 256, BIG_TILE_SIZE_N, BIG_TILE_SIZE_K, M_SWIZZLE, Operation, false, _T1, _T0>
                  <<<grid_dim, block_dim, 0, stream>>>(buf_b, buf_a, buf_c, K, N, types_match); });
      }
    }
  }
};

// PATTERN_BROADCAST_1
template <typename Operation, class _T0, class _T1>
struct BinaryOperationPattern<3, Operation, _T0, _T1>
{
  static void apply(torch::Tensor &input, torch::Tensor &other, torch::Tensor &output, bool order_flag)
  {
    int dim = input.dim();
    auto shape = output.sizes().vec();
    void *buf_a = reinterpret_cast<void *>(input.data_ptr());
    void *buf_b = reinterpret_cast<void *>(other.data_ptr());
    void *buf_c = reinterpret_cast<void *>(output.data_ptr());

    int num_elements = output.numel();
    int rem_dim_size = num_elements / (shape[dim - 2] * shape[dim - 1]);
    int M = dim == 2 ? 1 : rem_dim_size;
    int N = shape[dim - 2];
    int K = shape[dim - 1];

    constexpr uint32_t BIG_TILE_SIZE_N = 64;
    constexpr uint32_t BIG_TILE_SIZE_K = 64;
    constexpr uint32_t M_SWIZZLE = 8;
    const int grid_x = M * ((N + BIG_TILE_SIZE_N - 1) / BIG_TILE_SIZE_N) * ((K + BIG_TILE_SIZE_K - 1) / BIG_TILE_SIZE_K);
    const dim3 grid_dim(grid_x, 1, 1);
    const dim3 block_dim(256, 1, 1);
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    bool types_match = typeid(_T0) == typeid(_T1);

    if (order_flag)
    {
      VLLM_DISPATCH_FLOATING_TYPES(
          output.scalar_type(), "operator_bcast1_big_tile_kernel", [&]
          { aiter::operator_bcast1_big_tile_kernel<scalar_t, 256, BIG_TILE_SIZE_N, BIG_TILE_SIZE_K, M_SWIZZLE, Operation, true, _T0, _T1>
                <<<grid_dim, block_dim, 0, stream>>>(buf_a, buf_b, buf_c, K, N, types_match); });
    }
    else
    {
      VLLM_DISPATCH_FLOATING_TYPES(
          output.scalar_type(), "operator_bcast1_big_tile_kernel", [&]
          { aiter::operator_bcast1_big_tile_kernel<scalar_t, 256, BIG_TILE_SIZE_N, BIG_TILE_SIZE_K, M_SWIZZLE, Operation, false, _T1, _T0>
                <<<grid_dim, block_dim, 0, stream>>>(buf_b, buf_a, buf_c, K, N, types_match); });
    }
  }
};

// PATTERN_CONTIGUOUS
template <typename Operation, class _T0, class _T1>
struct BinaryOperationPattern<4, Operation, _T0, _T1>
{
  static void apply(torch::Tensor &input, torch::Tensor &other, torch::Tensor &output, bool order_flag)
  {
    int dim = input.dim();
    auto shape = output.sizes().vec();

    const uint32_t rows = 8;
    void *buf_a = reinterpret_cast<void *>(input.data_ptr());
    void *buf_b = reinterpret_cast<void *>(other.data_ptr());
    void *buf_c = reinterpret_cast<void *>(output.data_ptr());
    int num_elements = output.numel();
    int rem_dim_size = 1;
    int M, N, K;
    if (dim == 1)
    {
      M = 1;
      N = input.numel() / 128;
      K = 128;
    }
    else
    {
      for (int i = 0; i < dim - 2; ++i)
      {
        rem_dim_size *= shape[i];
      }
      M = dim == 3 ? shape[0] : rem_dim_size;
      N = shape[dim - 2];
      K = shape[dim - 1];
      if (N < rows)
      {
        K = N * K;
        N = M;
        M = 1;
      }
    }

    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    bool types_match = typeid(_T0) == typeid(_T1);
    int vec = 16 / output.element_size();
    hipDevice_t dev;
    hipDeviceProp_t dev_prop;
    hipGetDevice(&dev);
    hipGetDeviceProperties(&dev_prop, dev);
    uint32_t num_cu = dev_prop.multiProcessorCount;

    if (num_elements % vec == 0 && num_elements < num_cu * 256 * vec)
    {
      constexpr uint32_t wg = 256;
      const int grid_x = (num_elements / vec + wg - 1) / wg;
      const dim3 grid_dim(grid_x, 1, 1);
      const dim3 block_dim(wg, 1, 1);
      VLLM_DISPATCH_FLOATING_TYPES(
          output.scalar_type(), "operator_element_kernel", [&]
          { aiter::operator_element_kernel<scalar_t, Operation, _T0, _T1>
                <<<grid_dim, block_dim, 0, stream>>>(buf_a, buf_b, buf_c, num_elements, types_match); });
    }
    else if (N % rows == 0 && K % vec == 0)
    {
      constexpr uint32_t wg = 256;
      const int grid_x = (num_elements / (rows * vec) + wg - 1) / wg;
      const dim3 grid_dim(grid_x, 1, 1);
      const dim3 block_dim(wg, 1, 1);

      VLLM_DISPATCH_FLOATING_TYPES(
          output.scalar_type(), "operator_contiguous_kernel", [&]
          { aiter::operator_contiguous_kernel<scalar_t, rows, Operation, true, _T0, _T1>
                <<<grid_dim, block_dim, 0, stream>>>(buf_a, buf_b, buf_c, M, N, K, types_match); });
    }
    else
    {
      constexpr uint32_t wg = 256;
      constexpr uint32_t BIG_TILE_SIZE_N = 64;
      constexpr uint32_t BIG_TILE_SIZE_K = 64;
      constexpr uint32_t M_SWIZZLE = 8;
      const int grid_x = M * ((N + BIG_TILE_SIZE_N - 1) / BIG_TILE_SIZE_N) * ((K + BIG_TILE_SIZE_K - 1) / BIG_TILE_SIZE_K);
      const dim3 grid_dim(grid_x, 1, 1);
      const dim3 block_dim(wg, 1, 1);

      VLLM_DISPATCH_FLOATING_TYPES(
          output.scalar_type(), "operator_contiguous_big_tile_kernel", [&]
          { aiter::operator_contiguous_big_tile_kernel<scalar_t, 256, BIG_TILE_SIZE_N, BIG_TILE_SIZE_K, M_SWIZZLE, Operation, true, _T0, _T1>
                <<<grid_dim, block_dim, 0, stream>>>(buf_a, buf_b, buf_c, K, N, types_match); });
    }
  }
};

template <int pattern, typename Operation, class _T0, class _T1>
void binary_operation_process(torch::Tensor &input, torch::Tensor &other, torch::Tensor &output, bool order_flag)
{
  BinaryOperationPattern<pattern, Operation, _T0, _T1>::apply(input, other, output, order_flag);
}

#define DISPATCH_SECOND(pattern, Operation, _T0, scalar_type, cpp_type)                            \
  case scalar_type:                                                                                \
    binary_operation_process<pattern, Operation, _T0, cpp_type>(input, other, output, order_flag); \
    break

#define DISPATCH_FIRST(pattern, Operation, scalar_type, cpp_type)                    \
  case scalar_type:                                                                  \
    dispatch_second<pattern, Operation, cpp_type>(input, other, output, order_flag); \
    break

template <int pattern, typename Operation, typename _T0>
void dispatch_second(torch::Tensor &input, torch::Tensor &other, torch::Tensor &output, bool order_flag)
{
  switch (other.scalar_type())
  {
    DISPATCH_SECOND(pattern, Operation, _T0, torch::kFloat32, float);
    DISPATCH_SECOND(pattern, Operation, _T0, torch::kFloat64, double);
    DISPATCH_SECOND(pattern, Operation, _T0, torch::kInt32, int);
    DISPATCH_SECOND(pattern, Operation, _T0, torch::kInt64, long long);
    DISPATCH_SECOND(pattern, Operation, _T0, torch::kBool, bool);
    DISPATCH_SECOND(pattern, Operation, _T0, torch::kHalf, torch::Half);
    DISPATCH_SECOND(pattern, Operation, _T0, torch::kBFloat16, torch::BFloat16);
  default:
    break;
  }
}

template <int pattern, typename Operation>
void dispatch_first(torch::Tensor &input, torch::Tensor &other, torch::Tensor &output, bool order_flag)
{
  switch (input.scalar_type())
  {
    DISPATCH_FIRST(pattern, Operation, torch::kFloat32, float);
    DISPATCH_FIRST(pattern, Operation, torch::kFloat64, double);
    DISPATCH_FIRST(pattern, Operation, torch::kInt32, int);
    DISPATCH_FIRST(pattern, Operation, torch::kInt64, long long);
    DISPATCH_FIRST(pattern, Operation, torch::kBool, bool);
    DISPATCH_FIRST(pattern, Operation, torch::kHalf, torch::Half);
    DISPATCH_FIRST(pattern, Operation, torch::kBFloat16, torch::BFloat16);
  default:
    break;
  }
}

#undef DISPATCH_SECOND
#undef DISPATCH_FIRST

template <typename Operation, bool Inplace = false>
torch::Tensor binary_operation(torch::Tensor &input, torch::Tensor &other)
{
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  int dim = input.dim();

  bool is_support = false;
  bool order_flag = true;
  int pattern = 0;
  constexpr uint32_t PATTERN_TRANSPOSE = 1;
  constexpr uint32_t PATTERN_BROADCAST_0 = 2;
  constexpr uint32_t PATTERN_BROADCAST_1 = 3;
  constexpr uint32_t PATTERN_CONTIGUOUS = 4;

  if (!is_support)
  {
    is_support = true;
    is_support &= (input.dim() == other.dim());
    is_support &= input.is_contiguous() == other.is_contiguous();
    is_support &= input.is_contiguous() == true;
    if (input.dim() == 1)
    {
      is_support &= input.numel() % 128 == 0;
    }
    for (int i = 0; i < input.dim() && is_support; ++i)
    {
      is_support &= (input.size(i) == other.size(i));
    }
    pattern = is_support ? PATTERN_CONTIGUOUS : 0;
  }

  if (!is_support && dim == 3)
  {
    if (input.is_contiguous() != other.is_contiguous())
    {
      auto tensor_not_conti = input.is_contiguous() ? other : input;
      order_flag = !input.is_contiguous() ? true : false;
      is_support = true;
      // avoid broadcast
      is_support &= input.dim() == other.dim();
      is_support &= input.size(0) == other.size(0);
      is_support &= input.size(1) == other.size(1);
      is_support &= input.size(2) == other.size(2);
      is_support &= tensor_not_conti.stride(1) == 1;
      pattern = is_support ? PATTERN_TRANSPOSE : 0;
    }
    else if (input.is_contiguous() && other.is_contiguous())
    {
      is_support = false;

      if (!is_support && other.size(0) == 1)
      {
        is_support = true;
        is_support &= input.dim() == other.dim();
        is_support &= input.size(0) > 1;
        is_support &= input.size(1) == other.size(1);
        is_support &= input.size(2) == other.size(2);
        pattern = is_support ? PATTERN_BROADCAST_0 : 0;
        order_flag = true;
      }

      if (!is_support && input.size(0) == 1)
      {
        is_support = true;
        is_support &= input.dim() == other.dim();
        is_support &= other.size(0) > 1;
        is_support &= input.size(1) == other.size(1);
        is_support &= input.size(2) == other.size(2);
        pattern = is_support ? PATTERN_BROADCAST_0 : 0;
        order_flag = false;
      }

      if (!is_support && input.size(1) == 1)
      {
        is_support = true;
        is_support &= input.dim() == other.dim();
        is_support &= other.size(1) > 1;
        is_support &= input.size(0) == other.size(0);
        is_support &= input.size(2) == other.size(2);
        pattern = is_support ? PATTERN_BROADCAST_1 : 0;
        order_flag = true;
      }

      if (!is_support && other.size(1) == 1)
      {
        is_support = true;
        is_support &= input.dim() == other.dim();
        is_support &= input.size(1) > 1;
        is_support &= input.size(0) == other.size(0);
        is_support &= input.size(2) == other.size(2);
        pattern = is_support ? PATTERN_BROADCAST_1 : 0;
        order_flag = false;
      }
    }
  }

  if (is_support)
  {
    auto in0_dtype = input.dtype();
    auto in1_dtype = other.dtype();
    torch::ScalarType out_dtype = torch::promote_types(input.scalar_type(), other.scalar_type());
    std::vector<int64_t> out_shape = broadcastShapes(input, other);
    auto device = input.device();
    auto options = torch::TensorOptions().dtype(out_dtype).device(input.device());

    torch::Tensor output;
    if constexpr(Inplace)
    {
      output = input;
    }
    else
    {
      output = torch::empty(out_shape, options);
    }

    if (pattern == PATTERN_TRANSPOSE)
    {
      dispatch_first<1, Operation>(input, other, output, order_flag);
    }
    else if (pattern == PATTERN_BROADCAST_0)
    {
      dispatch_first<2, Operation>(input, other, output, order_flag);
    }
    else if (pattern == PATTERN_BROADCAST_1)
    {
      dispatch_first<3, Operation>(input, other, output, order_flag);
    }
    else if (pattern == PATTERN_CONTIGUOUS)
    {
      dispatch_first<4, Operation>(input, other, output, order_flag);
    }
    return output;
  }
  else
  {
    return aiter::aten_compute<Operation>(input, other);
  }
}

torch::Tensor aiter_add(torch::Tensor &input, torch::Tensor &other)
{
  return binary_operation<aiter::AddOp, false>(input, other);
}

torch::Tensor aiter_sub(torch::Tensor &input, torch::Tensor &other)
{
  return binary_operation<aiter::SubOp, false>(input, other);
}

torch::Tensor aiter_mul(torch::Tensor &input, torch::Tensor &other)
{
  return binary_operation<aiter::MulOp, false>(input, other);
}

torch::Tensor aiter_div(torch::Tensor &input, torch::Tensor &other)
{
  return binary_operation<aiter::DivOp, false>(input, other);
}

// inp interface
torch::Tensor aiter_add_(torch::Tensor &input, torch::Tensor &other)
{
  return binary_operation<aiter::AddOp, true>(input, other);
}

torch::Tensor aiter_sub_(torch::Tensor &input, torch::Tensor &other)
{
  return binary_operation<aiter::SubOp, true>(input, other);
}

torch::Tensor aiter_mul_(torch::Tensor &input, torch::Tensor &other)
{
  return binary_operation<aiter::MulOp, true>(input, other);
}

torch::Tensor aiter_div_(torch::Tensor &input, torch::Tensor &other)
{
  return binary_operation<aiter::DivOp, true>(input, other);
}
