#include "hip/hip_runtime.h"
// SPDX-License-Identifier: MIT
// Copyright (c) 2024, Advanced Micro Devices, Inc. All rights reserved.
/*
 * @Script: topk_softmax_kernels_group.cu
 * @Author: valarLip
 * @Email: lingpeng.jin@amd.com
 * @Create At: 2025-03-01 12:16:14
 * @Last Modified By: valarLip
 * @Last Modified At: 2025-03-04 22:27:28
 * @Description: This is description.
 */

#include <hip/hip_runtime.h>
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include "dispatch_utils.h"

#define WARP_SIZE 64
namespace aiter
{
    template <typename T, typename F>
    __device__ constexpr T wave_reduce(T local, F reduce_f)
    {
      constexpr int reduce_stage = 6; // 1<<6=64
      T v_local = local;
#pragma unroll
      for (int i_stage = 0; i_stage < reduce_stage; i_stage++)
      {
        int src_lane = __lane_id() ^ (1 << i_stage);
        int32_t v_remote_tmp =
            __builtin_amdgcn_ds_bpermute(src_lane << 2, __builtin_bit_cast(int32_t, v_local));
        T v_remote = __builtin_bit_cast(T, v_remote_tmp);
        v_local = reduce_f(v_local, v_remote);
      }
      return v_local;
    }

    __inline__ __device__ void warpReduceMax(float &val, int &idx)
    {
        static_assert(64 == WARP_SIZE, "WARP_SIZE == 64");
#pragma unroll
        for (int i = 0; i < 6; i++)
        {
            int offset = 1 << i;
            float tmp_val = __shfl_down(val, offset);
            int tmp_idx = __shfl_down(idx, offset);
            if (tmp_val > val)
            {
                val = tmp_val;
                idx = tmp_idx;
            }
        }
    }

    __device__ void blockReduceMax(float &val, int &idx)
    {
        __shared__ float shared_vals[32];
        __shared__ int shared_idxs[32];

        int lane = threadIdx.x % WARP_SIZE;
        int wid = threadIdx.x / WARP_SIZE;

        warpReduceMax(val, idx);

        if (lane == 0)
        {
            shared_vals[wid] = val;
            shared_idxs[wid] = idx;
        }
        __syncthreads();

        if (wid == 0)
        {
            val = (lane < (blockDim.x + WARP_SIZE - 1) / WARP_SIZE) ? shared_vals[lane] : -INFINITY;
            idx = (lane < (blockDim.x + WARP_SIZE - 1) / WARP_SIZE) ? shared_idxs[lane] : -1;

            warpReduceMax(val, idx);
        }
        __syncthreads();
    }

    template <typename DTYPE_I, typename fvec, int NUM_GRP, bool need_renorm, bool isBiased, bool isSoftmax>
    __global__ void grouped_topk_kernel(
        const DTYPE_I *__restrict__ gating_output,   // [num_tokens, hidden_size]
        const DTYPE_I *__restrict__ correction_bias, // [num_expert]
        float *__restrict__ topk_weights,            // [num_tokens, topk]
        int *__restrict__ topk_ids,                  // [num_tokens, topk]
        const size_t stride_tk,
        const int num_experts,
        const int topk,
        const int topk_group,
        const int num_tokens,
        const float routed_scaling_factor)
    {
        static_assert(NUM_GRP <= WARP_SIZE, "NUM_GRP must be <= WARP_SIZE");
        // 256 E, 8->4 group, 32 e/group
        const int experts_per_group = num_experts / NUM_GRP;
        extern __shared__ char shared_mem[];
        const int token_idx = blockIdx.x;

        char *ptr = (char *)(((size_t)shared_mem + 255) & ~255);
        float *scores = reinterpret_cast<float *>(ptr);
        ptr += num_experts * sizeof(float);

        float *group_scores = reinterpret_cast<float *>(ptr);
        ptr += NUM_GRP * sizeof(float);

        bool *group_mask = reinterpret_cast<bool *>(ptr);
        ptr += NUM_GRP * sizeof(bool);

        int *topk_indices = reinterpret_cast<int *>(ptr);
        ptr += topk * sizeof(int);

        float *topk_values = reinterpret_cast<float *>(ptr);
        // ptr += topk * sizeof(float);

        // int *topk_indices_f = reinterpret_cast<int *>(ptr);
        // ptr += topk * sizeof(int);

        // float *topk_values_f = reinterpret_cast<float *>(ptr);

        fvec *scores_vec = reinterpret_cast<fvec *>(scores);
        constexpr uint32_t vec_size = sizeof(fvec) / sizeof(float);

        if constexpr (!isSoftmax)
        {
            for (int e = threadIdx.x; e < num_experts; e += blockDim.x)
            {
                float gating = static_cast<float>(gating_output[token_idx * num_experts + e]);
                gating = 1.0f / (1.0f + expf(-gating));
                if constexpr (isBiased)
                {
                    gating += correction_bias[e];
                }
                scores[e] = gating;
            }
            __syncthreads();
        }
        else
        {
            __shared__ float sdata;
            float max_val = -INFINITY;
            for (int e = threadIdx.x; e < num_experts; e += blockDim.x)
            {

                float gating = gating_output[token_idx * num_experts + e];
                scores[e] = gating;
                if (gating > max_val)
                {
                    max_val = gating;
                }
            }
            __syncthreads();
#pragma unroll
            for (int i = 0; i < 6; i++)
            {
                int offset = 1 << i;
                float tmp_val = __shfl_down(max_val, offset);
                if (tmp_val > max_val)
                {
                    max_val = tmp_val;
                }
            }
            if (threadIdx.x == 0)
            {
                sdata = max_val;
            }
            __syncthreads();
            max_val = sdata;
            float thread_sum = 0.0;
            for (int e = threadIdx.x; e < num_experts; e += blockDim.x)
            {
                scores[e] = expf(scores[e] - max_val);
                thread_sum += scores[e];
            }
            __syncthreads();
            thread_sum = wave_reduce(thread_sum, [](float a, float b) { return a + b; });
            for (int e = threadIdx.x; e < num_experts; e += blockDim.x)
            {
                scores[e] /= thread_sum;
            }
            __syncthreads();
        }

        if constexpr (isBiased)
        {
#pragma unroll
            for (int g = threadIdx.x; g < NUM_GRP; g += blockDim.x)
            {
                float max1 = -INFINITY, max2 = -INFINITY;
                const int start = g * experts_per_group;
                const int end = start + experts_per_group;

                for (int e = start; e < end; ++e)
                {
                    if (scores[e] > max1)
                    {
                        max2 = max1;
                        max1 = scores[e];
                    }
                    else if (scores[e] > max2)
                    {
                        max2 = scores[e];
                    }
                }
                group_scores[g] = max1 + max2;
                group_mask[g] = false;
            }
            __syncthreads();
        }
        else
        {
#pragma unroll
            for (int g = threadIdx.x; g < NUM_GRP; g += blockDim.x)
            {
                float max1 = -INFINITY;
                const int start = g * experts_per_group;
                const int end = start + experts_per_group;
                for (int e = start; e < end; ++e)
                {
                    if (scores[e] > max1)
                    {
                        max1 = scores[e];
                    }
                }
                group_scores[g] = max1;
                group_mask[g] = false;
            }
            __syncthreads();
        }

        for (int k = 0; k < topk_group; k++)
        {
            float max_val = -INFINITY;
            int max_idx = -1;
#pragma unroll
            for (int g = threadIdx.x; g < NUM_GRP; g += blockDim.x)
            {
                if (group_scores[g] > max_val)
                {
                    max_val = group_scores[g];
                    max_idx = g;
                }
            }
            warpReduceMax(max_val, max_idx);
            if (threadIdx.x == 0 && max_idx != -1)
            {
                group_mask[max_idx] = true;
                group_scores[max_idx] = -INFINITY;
            }
            __syncthreads();
        }

        // lip: TODO we can do vec here if experts_per_group%vec_size==0
        for (int e = threadIdx.x; e < num_experts; e += blockDim.x)
        {
            int group_idx = e / experts_per_group;
            if (!group_mask[group_idx])
            {
                scores[e] = -INFINITY;
            }
        }
        __syncthreads();

        float sum = 0.0f;
        for (int k = 0; k < topk; ++k)
        {
            float max_val = -INFINITY;
            int max_idx = -1;

            for (int e = threadIdx.x; e < num_experts / vec_size; e += blockDim.x)
            {
                union
                {
                    fvec vec;
                    float f[vec_size];
                } tmp;
                tmp.vec = scores_vec[e];
#pragma unroll
                for (size_t i = 0; i < vec_size; i++)
                {
                    if (tmp.f[i] > max_val)
                    {
                        max_val = tmp.f[i];
                        max_idx = e * vec_size + i;
                    }
                }
            }

            warpReduceMax(max_val, max_idx);
            // blockReduceMax(max_val, max_idx);

            if (threadIdx.x == 0)
            {
                if (max_idx == -1)
                {
                    max_idx = k;
                    max_val = scores[max_idx];
                }
                if constexpr (isBiased)
                {
                    max_val -= correction_bias[max_idx];
                }
                scores[max_idx] = -INFINITY;
                topk_indices[k] = max_idx;
                topk_values[k] = max_val;
                if (need_renorm)
                {
                    sum += max_val;
                }
            }
            __syncthreads();
        }

        if (need_renorm)
        {
            if (threadIdx.x == 0)
            {
                scores[0] = routed_scaling_factor / sum; // reuse lds
            }
            __syncthreads();
            sum = scores[0];
        }
        else
        {
            sum = routed_scaling_factor;
        }

        for (int k = threadIdx.x; k < topk; k += blockDim.x)
        {
            topk_weights[token_idx * stride_tk + k] = topk_values[k] * sum;
            topk_ids[token_idx * stride_tk + k] = topk_indices[k];
        }
    }
} // namespace aiter end

#define LAUNCH_KERNEL()      \
    switch (num_experts % 4) \
    {                        \
    case 0:                  \
        LAUNCHER2(float4)    \
        break;               \
    case 2:                  \
        LAUNCHER2(float2)    \
        break;               \
    default:                 \
        LAUNCHER2(float)     \
        break;               \
    }
#define LAUNCHER2(VEC_F)                                                        \
    switch (num_expert_group)                                                   \
    {                                                                           \
    case 8:                                                                     \
        LAUNCHER3(VEC_F, 8)                                                     \
        break;                                                                  \
    case 4:                                                                     \
        LAUNCHER3(VEC_F, 4)                                                     \
        break;                                                                  \
    case 2:                                                                     \
        LAUNCHER3(VEC_F, 2)                                                     \
        break;                                                                  \
    case 1:                                                                     \
        LAUNCHER3(VEC_F, 1)                                                     \
        break;                                                                  \
    default:                                                                    \
        TORCH_CHECK(false, "Unsupported num_expert_group: ", num_expert_group); \
        break;                                                                  \
    }
#define LAUNCHER3(VEC_F, NUM_GRP)        \
    switch (need_renorm)                 \
    {                                    \
    case true:                           \
        LAUNCHER4(VEC_F, NUM_GRP, true)  \
        break;                           \
    default:                             \
        LAUNCHER4(VEC_F, NUM_GRP, false) \
    }

#define LAUNCHER4(VEC_F, NUM_GRP, need_renorm)                                        \
    if constexpr (isBiased)                                                           \
    {                                                                                 \
        LAUNCHER_biased_grouped_topk_kernel(VEC_F, NUM_GRP, need_renorm, true, false) \
    }                                                                                 \
    else                                                                              \
    {                                                                                 \
        if (isSoftmax)                                                                \
        {                                                                             \
            LAUNCHER_grouped_topk_kernel(VEC_F, NUM_GRP, need_renorm, false, true)    \
        }                                                                             \
        else                                                                          \
        {                                                                             \
            LAUNCHER_grouped_topk_kernel(VEC_F, NUM_GRP, need_renorm, false, false)   \
        }                                                                             \
    }

#define LAUNCHER_biased_grouped_topk_kernel(VEC_F, NUM_GRP, need_renorm, isBiased, isSoftmax)    \
    VLLM_DISPATCH_FLOATING_TYPES(                                                                \
        gating_output.scalar_type(), "biased_grouped_topk_kernel", [&]                           \
        { aiter::grouped_topk_kernel<scalar_t, VEC_F, NUM_GRP, need_renorm, isBiased, isSoftmax> \
              <<<grid, block, shared_mem_size, stream>>>(                                        \
                  gating_output.data_ptr<scalar_t>(),                                            \
                  correction_bias.data_ptr<scalar_t>(),                                          \
                  topk_weights.data_ptr<float>(),                                                \
                  topk_ids.data_ptr<int>(),                                                      \
                  stride_tk,                                                                     \
                  num_experts,                                                                   \
                  topk,                                                                          \
                  topk_grp, num_tokens, routed_scaling_factor); });

#define LAUNCHER_grouped_topk_kernel(VEC_F, NUM_GRP, need_renorm, isBiased, isSoftmax)           \
    VLLM_DISPATCH_FLOATING_TYPES(                                                                \
        gating_output.scalar_type(), "grouped_topk_kernel", [&]                                  \
        { aiter::grouped_topk_kernel<scalar_t, VEC_F, NUM_GRP, need_renorm, isBiased, isSoftmax> \
              <<<grid, block, shared_mem_size, stream>>>(                                        \
                  gating_output.data_ptr<scalar_t>(),                                            \
                  nullptr,                                                                       \
                  topk_weights.data_ptr<float>(),                                                \
                  topk_ids.data_ptr<int>(),                                                      \
                  stride_tk,                                                                     \
                  num_experts,                                                                   \
                  topk,                                                                          \
                  topk_grp, num_tokens, routed_scaling_factor); });

void biased_grouped_topk(
    torch::Tensor &gating_output,   // [num_tokens, num_experts]
    torch::Tensor &correction_bias, // [num_expert]
    torch::Tensor &topk_weights,    // [num_tokens, topk]
    torch::Tensor &topk_ids,        // [num_tokens, topk]
    int num_expert_group,
    int topk_grp,
    bool need_renorm,
    const float routed_scaling_factor=1.)
{
    const bool isBiased = true;
    bool isSoftmax = false;
    int num_tokens = gating_output.size(0);
    int num_experts = gating_output.size(1);
    int topk = topk_ids.size(1);
    size_t stride_tk = topk_ids.stride(0);
    TORCH_CHECK(stride_tk == topk_weights.stride(0), "topk_ids.stride(0) == topk_weights.stride(0)");
    TORCH_CHECK(gating_output.dtype() == correction_bias.dtype(), "gating_output.dtype() == correction_bias.dtype()");

    dim3 grid(num_tokens);
    dim3 block(64);
    size_t shared_mem_size = (num_experts * sizeof(float) +
                              num_expert_group * sizeof(float) +
                              num_expert_group * sizeof(bool) +
                              topk * sizeof(int) +
                              topk * sizeof(float) + 255) &
                             ~255;

    const at::cuda::OptionalCUDAGuard device_guard(device_of(gating_output));
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    LAUNCH_KERNEL()
}

void grouped_topk(
    torch::Tensor &gating_output, // [num_tokens, num_experts]
    torch::Tensor &topk_weights,  // [num_tokens, topk]
    torch::Tensor &topk_ids,      // [num_tokens, topk]
    int num_expert_group,
    int topk_grp,
    bool need_renorm,
    std::string scoring_func = "softmax",
    const float routed_scaling_factor = 1.)
{
    TORCH_CHECK((scoring_func == "softmax") || (scoring_func == "sigmoid"), "grouped_topk scoring_func only suppot softmax or sigmoid");
    const bool isBiased = false;
    bool isSoftmax = scoring_func == "softmax" ? true : false;
    int num_tokens = gating_output.size(0);
    int num_experts = gating_output.size(1);
    int topk = topk_ids.size(1);
    size_t stride_tk = topk_ids.stride(0);
    auto correction_bias = topk_ids;
    TORCH_CHECK(stride_tk == topk_weights.stride(0), "topk_ids.stride(0) == topk_weights.stride(0)");

    dim3 grid(num_tokens);
    dim3 block(64);
    size_t shared_mem_size = (num_experts * sizeof(float) +
                              num_expert_group * sizeof(float) +
                              num_expert_group * sizeof(bool) +
                              topk * sizeof(int) +
                              topk * sizeof(float) + 255) &
                             ~255;

    const at::cuda::OptionalCUDAGuard device_guard(device_of(gating_output));
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    LAUNCH_KERNEL()
}

#undef LAUNCHER4
#undef LAUNCHER3
#undef LAUNCHER2
#undef LAUNCH_KERNEL