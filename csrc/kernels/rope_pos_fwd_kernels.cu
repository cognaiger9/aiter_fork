#include "hip/hip_runtime.h"
// SPDX-License-Identifier: MIT
// Copyright (c) 2025, Advanced Micro Devices, Inc. All rights reserved.

#include "rope_common.h"

// =====================================================================================================================
// Interfaces
//

void rope_cached_positions_2c_fwd_impl(
    torch::Tensor&       output_x,      // [s, b, h, d]
    torch::Tensor&       output_y,      // [s, b, h, d]
    const torch::Tensor& input_x,       // [s, b, h, d]
    const torch::Tensor& input_y,       // [s, b, h, d]
    const torch::Tensor& cos,           // [s, 1, 1, d // 2] if reuse_freqs_front_part else [s, 1, 1, d]
    const torch::Tensor& sin,           // [s, 1, 1, d // 2] if reuse_freqs_front_part else [s, 1, 1, d]
    const torch::Tensor& positions,     // [s, b]
    const int32_t        rotate_style,  // 0: NEOX style, 1: GPT-J style
    const bool           reuse_freqs_front_part,
    const bool           nope_first)
{
    // Get sizes of input and output
    const int32_t size_s   = min(input_x.size(0), positions.size(0));
    const int32_t size_b   = min(input_x.size(1), positions.size(1));
    const int32_t size_h_x = input_x.size(2);
    const int32_t size_h_y = input_y.size(2);
    const int32_t size_d   = input_x.size(3);
    const int32_t size_f   = cos.size(3);
    // Get strides of input
    const int32_t stride_ix_s = input_x.stride(0);
    const int32_t stride_ix_b = input_x.stride(1);
    const int32_t stride_ix_h = input_x.stride(2);
    const int32_t stride_ix_d = input_x.stride(3);
    const int32_t stride_iy_s = input_y.stride(0);
    const int32_t stride_iy_b = input_y.stride(1);
    const int32_t stride_iy_h = input_y.stride(2);
    const int32_t stride_iy_d = input_y.stride(3);
    // Get strides of output
    const int32_t stride_ox_s = output_x.stride(0);
    const int32_t stride_ox_b = output_x.stride(1);
    const int32_t stride_ox_h = output_x.stride(2);
    const int32_t stride_ox_d = output_x.stride(3);
    const int32_t stride_oy_s = output_y.stride(0);
    const int32_t stride_oy_b = output_y.stride(1);
    const int32_t stride_oy_h = output_y.stride(2);
    const int32_t stride_oy_d = output_y.stride(3);
    // Get strides of positions and offsets
    assert(1 == positions.stride(1) && 2 == positions.dim());

    DISPATCH_ROPE_TYPES_PARAMS(
        input_x.scalar_type(),
        cos.scalar_type(),
        rotate_style,
        reuse_freqs_front_part,
        nope_first,
        "dispatch_2c_sbhd_cached_indirect<OpCachedFwd, ...>",
        dispatch_2c_sbhd_cached_indirect<OpCachedFwd, RotateStyle, ReuseFreqsFrontPart, NopeFirst>(
            output_x.data_ptr<scalar_t_0>(),
            output_y.data_ptr<scalar_t_0>(),
            input_x.data_ptr<scalar_t_0>(),
            input_y.data_ptr<scalar_t_0>(),
            cos.data_ptr<scalar_t_1>(),
            sin.data_ptr<scalar_t_1>(),
            positions.data_ptr<int64_t>(),
            size_s, size_b, size_h_x, size_h_y, size_d,
            size_f, // size of last dimension of freqs.
            stride_ix_s, stride_ix_b, stride_ix_h, stride_ix_d,
            stride_iy_s, stride_iy_b, stride_iy_h, stride_iy_d,
            stride_ox_s, stride_ox_b, stride_ox_h, stride_ox_d,
            stride_oy_s, stride_oy_b, stride_oy_h, stride_oy_d););
}

void rope_cached_positions_offsets_2c_fwd_impl(
    torch::Tensor&       output_x,      // [s, b, h, d]
    torch::Tensor&       output_y,      // [s, b, h, d]
    const torch::Tensor& input_x,       // [s, b, h, d]
    const torch::Tensor& input_y,       // [s, b, h, d]
    const torch::Tensor& cos,           // [s, 1, 1, d // 2] if reuse_freqs_front_part else [s, 1, 1, d]
    const torch::Tensor& sin,           // [s, 1, 1, d // 2] if reuse_freqs_front_part else [s, 1, 1, d]
    const torch::Tensor& positions,     // [s, b]
    const torch::Tensor& offsets,       // [s, b]
    const int32_t        rotate_style,  // 0: NEOX style, 1: GPT-J style
    const bool           reuse_freqs_front_part,
    const bool           nope_first)
{
    // Get sizes of input and output
    const int32_t size_s   = min(min(input_x.size(0), positions.size(0)), offsets.size(0));
    const int32_t size_b   = min(min(input_x.size(1), positions.size(1)), offsets.size(1));
    const int32_t size_h_x = input_x.size(2);
    const int32_t size_h_y = input_y.size(2);
    const int32_t size_d   = input_x.size(3);
    const int32_t size_f   = cos.size(3);
    // Get strides of input
    const int32_t stride_ix_s = input_x.stride(0);
    const int32_t stride_ix_b = input_x.stride(1);
    const int32_t stride_ix_h = input_x.stride(2);
    const int32_t stride_ix_d = input_x.stride(3);
    const int32_t stride_iy_s = input_y.stride(0);
    const int32_t stride_iy_b = input_y.stride(1);
    const int32_t stride_iy_h = input_y.stride(2);
    const int32_t stride_iy_d = input_y.stride(3);
    // Get strides of output
    const int32_t stride_ox_s = output_x.stride(0);
    const int32_t stride_ox_b = output_x.stride(1);
    const int32_t stride_ox_h = output_x.stride(2);
    const int32_t stride_ox_d = output_x.stride(3);
    const int32_t stride_oy_s = output_y.stride(0);
    const int32_t stride_oy_b = output_y.stride(1);
    const int32_t stride_oy_h = output_y.stride(2);
    const int32_t stride_oy_d = output_y.stride(3);
    // Get strides of positions and offsets
    assert(1 == positions.stride(1) && 2 == positions.dim());
    assert(1 == offsets.stride(1)   && 2 == offsets.dim());

    DISPATCH_ROPE_TYPES_PARAMS(
        input_x.scalar_type(),
        cos.scalar_type(),
        rotate_style,
        reuse_freqs_front_part,
        nope_first,
        "dispatch_2c_sbhd_cached_indirect2<OpCachedFwd, ...>",
        dispatch_2c_sbhd_cached_indirect2<OpCachedFwd, RotateStyle, ReuseFreqsFrontPart, NopeFirst>(
            output_x.data_ptr<scalar_t_0>(),
            output_y.data_ptr<scalar_t_0>(),
            input_x.data_ptr<scalar_t_0>(),
            input_y.data_ptr<scalar_t_0>(),
            cos.data_ptr<scalar_t_1>(),
            sin.data_ptr<scalar_t_1>(),
            positions.data_ptr<int64_t>(),
            offsets.data_ptr<int64_t>(),
            size_s, size_b, size_h_x, size_h_y, size_d,
            size_f, // size of last dimension of freqs.
            stride_ix_s, stride_ix_b, stride_ix_h, stride_ix_d,
            stride_iy_s, stride_iy_b, stride_iy_h, stride_iy_d,
            stride_ox_s, stride_ox_b, stride_ox_h, stride_ox_d,
            stride_oy_s, stride_oy_b, stride_oy_h, stride_oy_d););
}